#include "hip/hip_runtime.h"
﻿#include <stdlib.h>
#include <stdio.h>

// multiply matrix
__global__ void multKernel(int* c, const int* a, const int* b) {
	// 2 dimensional index
	int x = threadIdx.x;
	int y = threadIdx.y;

	// Cij = Aik + Bkj
	for (int k = 0; k < (blockDim.x); k++) {
		c[x*5 + y] += (a[x*5 + k] * b[k*5 + y]);
	}
}

// print matrix
void printSquareMatrix(const int *matrix, const int WIDTH) {
	for (int i = 0; i < WIDTH; i++) {
		for (int j = 0; j < WIDTH; j++) {
			printf("%d\t", matrix[i*WIDTH + j]);
		}
		printf("\n");
	}
}

int main() {
	// host array
	const int WIDTH = 5;
	int a[WIDTH][WIDTH];
	int b[WIDTH][WIDTH];
	int c[WIDTH][WIDTH] = { 0 };
	for (int y = 0; y < WIDTH; y++) {
		for (int x = 0; x < WIDTH; x++) {
			a[y][x] = y + x;
			b[y][x] = y + x;
		}
	}

	// device array
	int* dev_a, * dev_b, * dev_c = 0;
	hipMalloc((void**)&dev_a, WIDTH * WIDTH * sizeof(int));
	hipMalloc((void**)&dev_b, WIDTH * WIDTH * sizeof(int));
	hipMalloc((void**)&dev_c, WIDTH * WIDTH * sizeof(int));

	// copy matrix from host to device
	hipMemcpy(dev_a, a, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice);

	// multiply Matrix
	dim3 DimBlock(WIDTH, WIDTH);
	multKernel << <1, DimBlock >> > (dev_c, dev_a, dev_b);

	// copy result matrix from device to host
	hipMemcpy(c, dev_c, WIDTH * WIDTH * sizeof(int), hipMemcpyDeviceToHost);

	// free memory space
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	// print result matrix
	printf("행렬 A:\n");
	printSquareMatrix(*a, WIDTH);
	printf("\n행렬 B:\n");
	printSquareMatrix(*b, WIDTH);
	printf("\nA와 B의 행렬곱 결과행렬 C:\n");
	printSquareMatrix(*c, WIDTH);

	return 0;
}
